#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_tick.h"
#include <stdio.h>

__global__ void tickKernel(int *xArray, int *yArray, float *destXarray, float *destYarray, float *destRarray, int *destReached)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  float diffX = destXarray[i] - xArray[i];
  float diffY = destYarray[i] - yArray[i];

  float length = sqrt(diffX*diffX + diffY*diffY);
  destReached[i] = length < destRarray[i];

  xArray[i] = (int) round(xArray[i] + diffX/length);
  yArray[i] = (int) round(yArray[i] + diffY/length);
}

// Calculates and updates x/y positions, checks if agent has reached destination -> destReached
hipError_t tickCuda(int *xArray, int *yArray, float *destXarray, float *destYarray, float *destRarray, int *destReached, int NUM_BLOCKS, int THREADS_PER_BLOCK)
{
  hipError_t cudaStatus;
  int *dev_xArray;
  int *dev_yArray;
  float *dev_destXarray;
  float *dev_destYarray;
  float *dev_destRarray;
  int *dev_destReached;
  int size = NUM_BLOCKS * THREADS_PER_BLOCK;
  
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
    goto Error;
  }

  // Allocate GPU buffers for vectors
  cudaStatus = hipMalloc((void**)&dev_xArray, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void**)&dev_yArray, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void**)&dev_destXarray, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void**)&dev_destYarray, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void**)&dev_destRarray, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void**)&dev_destReached, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}

  // Copy input vectors from host memory to GPU buffers
  cudaStatus = hipMemcpy(dev_xArray, xArray, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_yArray, yArray, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_destXarray, destXarray, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_destYarray, destYarray, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_destRarray, destRarray, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_destReached, destReached, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}

  // Launch Kernel on the GPU with one thread for each element
  tickKernel <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (dev_xArray, dev_yArray, dev_destXarray, dev_destYarray, dev_destRarray, dev_destReached);
  
  // Check if kernel succeded
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "tickKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error;
  }

  // Copy data from device to host
  cudaStatus = hipMemcpy(xArray, dev_xArray, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(yArray, dev_yArray, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(destReached, dev_destReached, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
Error:
	/* hipFree(dev_xArray); */
	/* hipFree(dev_yArray); */
	/* hipFree(dev_destXarray); */
	/* hipFree(dev_destYarray); */
	/* hipFree(dev_destRarray); */
	/* hipFree(dev_destReached); */
	if (cudaStatus != 0){
		fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
	}
	/* else{ */
	/* 	fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing */
	/* } */

	return cudaStatus;
}
