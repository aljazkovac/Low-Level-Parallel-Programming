#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_heatmap.h"
#include <stdio.h>

__global__ void creationKernel(int *desiredX, int *desiredY, int *hm)
{
  //int i = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int x = desiredX[i];
  int y = desiredY[i];

  if (x < 0 || x >= SIZE || y < 0 || y >= SIZE) {}
  else {
    atomicAdd(&hm[y*SIZE + x], 40);
  }
}

__global__ void scalingKernel(int *hm, int *shm)
{
  __shared__ int shm_temp[16 * 16 * CELLSIZE * CELLSIZE];
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  int x_temp = threadIdx.x;
  int y_temp = threadIdx.y;

  int SMALL_SCALED_SIZE = 16 * CELLSIZE;

  hm[y*SIZE + x] = (int)round(hm[y*SIZE + x] * 0.80);
  hm[y*SIZE + x] = hm[y*SIZE + x] < 255 ? hm[y*SIZE + x] : 255;
  int value = hm[y*SIZE + x];
  for (int cellY = 0; cellY < CELLSIZE; cellY++)
    {
      for (int cellX = 0; cellX < CELLSIZE; cellX++)
	{
	  shm_temp[(y_temp * CELLSIZE + cellY)*SMALL_SCALED_SIZE + (x_temp * CELLSIZE + cellX)] = value;
	}
    }
  __syncthreads();
  for (int cellY = 0; cellY < CELLSIZE; cellY++)
    {
      for (int cellX = 0; cellX < CELLSIZE; cellX++)
	{
	  shm[(y * CELLSIZE + cellY)*SCALED_SIZE + (x * CELLSIZE + cellX)] = shm_temp[(y_temp * CELLSIZE + cellY)*SMALL_SCALED_SIZE + (x_temp * CELLSIZE + cellX)];
	}
    }
}

__global__ void blurKernel(int *shm, int *bhm)
{
  const int w[5][5] = {
    { 1, 4, 7, 4, 1 },
    { 4, 16, 26, 16, 4 },
    { 7, 26, 41, 26, 7 },
    { 4, 16, 26, 16, 4 },
    { 1, 4, 7, 4, 1 }
  };

  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

#define WEIGHTSUM 273
  // Apply gaussian blurfilter

  if (x < 2 || x > SCALED_SIZE - 2) {
    return;
  }
  if (y < 2 || y > SCALED_SIZE - 2) {
    return;
  }
  
  int sum = 0;
  for (int k = -2; k < 3; k++)
    {
      for (int l = -2; l < 3; l++)
	{
	  // sum += w[2 + k][2 + l] * scaled_heatmap[i + k][j + l];
	  sum += w[2 + k][2 + l] * shm[(y + k)*SCALED_SIZE + (x + l)];
	}
    }
  int value = sum / WEIGHTSUM;
  bhm[y*SCALED_SIZE + x] = 0x00FF0000 | value << 24;
}

// Calculates and updates x/y positions, checks if agent has reached destination -> destReached
hipError_t updateHeatmapCuda(int *desiredX, int *desiredY, int *hm, int *shm, int *bhm, int agents_size)
{
  hipError_t cudaStatus;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;


  // int NUM_BLOCKS = 2048;
  // int THREADS_PER_BLOCK = 512;
  //  int size = NUM_BLOCKS * THREADS_PER_BLOCK;

  int *dev_desiredX;
  int *dev_desiredY;
  int *dev_hm;
  int *dev_shm;
  int *dev_bhm;

  dim3 threadsPerBlock(16,16); // 256 threads per block
  dim3 numBlocks(SIZE/threadsPerBlock.x, SIZE/threadsPerBlock.y);
  
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
    goto Error;
  }

  // Allocate GPU buffers for agents
  cudaStatus = hipMalloc((void **)&dev_desiredX, agents_size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "dx hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_desiredY, agents_size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_hm, SIZE * SIZE * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}

  // Copy input vectors from host memory to GPU buffers
  cudaStatus = hipMemcpy(dev_desiredX, desiredX, agents_size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy dx to device failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_desiredY, desiredY, agents_size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy dy to failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_hm, hm, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy hm to device failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_shm, shm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy shm to device failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_bhm, bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy shm to device failed!"); goto Error;}

  

  // Launch Kernel on the GPU with one thread for each element
  // Set 10 blocks for hugeScenario.xml
  hipEventRecord(start);
  creationKernel <<<10, agents_size/10>>> (dev_desiredX, dev_desiredY, dev_hm);
  hipEventRecord(stop);
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "creationKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("creationKernel ms: %f\n", milliseconds);

  // Synchronize
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching creationKernel!\n", cudaStatus); goto Error;
  }

  hipEventRecord(start);
  scalingKernel <<<numBlocks, threadsPerBlock>>> (dev_hm, dev_shm);
  hipEventRecord(stop);
  
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "scalingKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("scaleKernel ms: %f\n", milliseconds);

  // Synchronize
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scalingKernel!\n", cudaStatus); goto Error;
  }

  // Blur filter
  hipEventRecord(start);
  blurKernel <<<numBlocks, threadsPerBlock>>> (dev_shm, dev_bhm);
  hipEventRecord(stop);
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "blurKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("blurKernel ms: %f\n", milliseconds);

  // Copy data from device to host
  cudaStatus = hipMemcpy(hm, dev_hm, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy hm to host failed!"); goto Error;}
  cudaStatus = hipMemcpy(shm, dev_shm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy shm to host failed!"); goto Error;}
  cudaStatus = hipMemcpy(bhm, dev_bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy shm to host failed!"); goto Error;}

 Error:
  hipFree(dev_desiredX);
  hipFree(dev_desiredY);
  hipFree(dev_hm);
  hipFree(dev_shm);
  hipFree(dev_bhm);
  if (cudaStatus != 0){
    fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
  }
  /* else{ */
  /* 	fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing */
  /* } */

  return cudaStatus;
}

// hipError_t allocCuda(int size)
// {
//   hipError_t cudaStatus;

//   // Allocate GPU buffers for agents
//   cudaStatus = hipMalloc((void **)&dev_desiredX, size * sizeof(int));
//   if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
//   cudaStatus = hipMalloc((void **)&dev_desiredY, size * sizeof(int));
//   if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  // cudaStatus = hipMalloc((void **)&dev_hm, SIZE * sizeof(int));
  // if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
//   cudaStatus = hipMalloc((void **)&dev_shm, SIZE * sizeof(int));
//   if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  
//   Error:
//   /* hipFree(dev_desiredX); */
//   /* hipFree(dev_desiredY); */
//   /* hipFree(dev_hm); */
//   /* hipFree(dev_shm); */
//   /* hipFree(dev_destReached); */
//   if (cudaStatus != 0){
//     fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
//   }
//   /* else{ */
//   /* 	fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing */
//   /* } */

//   return cudaStatus;
// }
