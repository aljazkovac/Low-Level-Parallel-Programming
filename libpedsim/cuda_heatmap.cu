#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_heatmap.h"
#include <stdio.h>

__global__ void creationKernel(int *desiredX, int *desiredY, int **heatmap)
{
  int i = threadIdx.x;

  int x = desiredX[i];
  int y = desiredY[i];

  if (x < 0 || x >= SIZE || y < 0 || y >= SIZE) {}
  else {
    atomicAdd(&heatmap[y][x], 40);
  }
}

__global__ void scalingKernel(int **heatmap, int **scaled_heatmap)
{
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
  heatmap[y][x] = heatmap[y][x] < 255 ? heatmap[y][x] : 255;
  int value = heatmap[y][x];
  for (int cellY = 0; cellY < CELLSIZE; cellY++)
    {
      for (int cellX = 0; cellX < CELLSIZE; cellX++)
	{
	  scaled_heatmap[y * CELLSIZE + cellY][x * CELLSIZE + cellX] = value;
	}
    }
}

/* __global__ void blurKernel() */
/* { */
  
/* } */

// Calculates and updates x/y positions, checks if agent has reached destination -> destReached
hipError_t updateHeatmapCuda(int *desiredX, int *desiredY, int **heatmap, int **scaled_heatmap)
{
  int agents_size = agents.size();
  hipError_t cudaStatus;
  /* int *dev_desiredX; */
  /* int *dev_desiredY; */
  /* int **dev_heatmap; */
  /* int **dev_scaled_heatmap; */
  int NUM_BLOCKS = 2048;
  int THREADS_PER_BLOCK = 256;
  int size = NUM_BLOCKS * THREADS_PER_BLOCK;
  
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers
  cudaStatus = hipMemcpy(dev_desiredX, desiredX, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_desiredY, desiredY, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_heatmap, heatmap, SIZE * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(dev_scaled_heatmap, scaled_heatmap, SIZE * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}

  // Launch Kernel on the GPU with one thread for each element
  creationKernel <<<1, size>>> (dev_desiredX, dev_desiredX, dev_heatmap);
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "tickKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  // Synchronize
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error;
  }
  
  dim3 threadsPerBlock(16,16); // 256 threads per block
  dim3 numBlocks(SIZE/threadsPerBlock.x, SIZE/threadsPerBlock.y);
  checkAndScaleKernel <<<numBlocks, threadsPerBlock>>> (dev_heatmap, dev_scaled_heatmap);
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {fprintf(stderr, "tickKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error;}

  // Synchronize
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error;
  }

  // Copy data from device to host
  cudaStatus = hipMemcpy(desiredX, dev_desiredX, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(desiredY, dev_desiredY, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(heatmap, dev_heatmap, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}
  cudaStatus = hipMemcpy(heatmap, dev_scaled_heatmap, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!"); goto Error;}

 Error:
  /* hipFree(dev_desiredX); */
  /* hipFree(dev_desiredY); */
  /* hipFree(dev_heatmap); */
  /* hipFree(dev_scaled_heatmap); */
  /* hipFree(dev_destReached); */
  if (cudaStatus != 0){
    fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
  }
  /* else{ */
  /* 	fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing */
  /* } */

  return cudaStatus;
}

hipError_t allocCuda(int size);
{
  // Allocate GPU buffers for agents
  cudaStatus = hipMalloc((void **)&dev_desiredX, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_desiredY, size * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_heatmap, SIZE * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  cudaStatus = hipMalloc((void **)&dev_scaled_heatmap, SIZE * sizeof(int));
  if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!"); goto Error;}
  
  Error:
  /* hipFree(dev_desiredX); */
  /* hipFree(dev_desiredY); */
  /* hipFree(dev_heatmap); */
  /* hipFree(dev_scaled_heatmap); */
  /* hipFree(dev_destReached); */
  if (cudaStatus != 0){
    fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
  }
  /* else{ */
  /* 	fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing */
  /* } */

  return cudaStatus;
}
